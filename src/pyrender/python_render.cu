// Copyright (C) 2025 Qianyue He
//
// This program is free software: you can redistribute it and/or
// modify it under the terms of the GNU Affero General Public License
// as published by the Free Software Foundation, either
// version 3 of the License, or (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See
// the GNU Affero General Public License for more details.
//
// You should have received a copy of the GNU Affero General
// Public License along with this program. If not, see
//
//             <https://www.gnu.org/licenses/>.

/**
 * @author Qianyue He
 * @brief Renderer Nanobind bindings
 * @date 2025.01.10
 */
#include "./python_render.cuh"
#include "core/scene.cuh"
#include "core/serialize.h"
#include "core/stats.h"
#include "renderer/bvh_cost.cuh"
#include "renderer/light_tracer.cuh"
#include "renderer/volume_pt.cuh"
#include "renderer/wf_path_tracer.cuh"

template <size_t Ndim>
static nb::ndarray<nb::pytorch, float>
gpu_ndarray_deep_copy(const float *gpu_src_ptr, size_t width, size_t height,
                      int dev_id = 0) {
    int num_elements = width * height * Ndim;

    float *gpu_dst_ptr;
    CUDA_CHECK_RETURN(
        hipMalloc((void **)&gpu_dst_ptr, num_elements * sizeof(float)));

    nb::capsule deleter(
        gpu_dst_ptr, [](void *p) noexcept { CUDA_CHECK_RETURN(hipFree(p)); });

    CUDA_CHECK_RETURN(hipMemcpy(gpu_dst_ptr, gpu_src_ptr,
                                 num_elements * sizeof(float),
                                 hipMemcpyDeviceToDevice));
    return nb::ndarray<nb::pytorch, float>(gpu_dst_ptr, {height, width, Ndim},
                                           deleter, {}, nb::dtype<float>(),
                                           nb::device::cuda::value, dev_id);
}

nb::ndarray<nb::pytorch, float> PythonRenderer::render() {
    TicTocLocal timer;
    const float *gpu_ptr =
        rdr->render_raw(scene->config.md, scene->config.gamma_correction);
    ftimer->record(timer.toc());
    return gpu_ndarray_deep_copy<4>(gpu_ptr, rdr->width(), rdr->height(),
                                    device_id);
}

nb::ndarray<nb::pytorch, float> PythonRenderer::variance() {
    const float *var_buffer = rdr->get_variance_buffer();
    if (var_buffer) {
        return gpu_ndarray_deep_copy<1>(var_buffer, rdr->width(), rdr->height(),
                                        device_id);
    }
    return {};
}

PythonRenderer::PythonRenderer(const nb::str &xml_path, int _device_id,
                               int seed_offset)
    : valid(true), device_id(_device_id) {
    CUDA_CHECK_RETURN(hipSetDevice(_device_id));
    CUDA_CHECK_RETURN(hipFree(nullptr)); // initialize CUDA

    std::string path = std::string(xml_path.c_str());
    ftimer = std::make_unique<SlidingWindowAverage>(32);
    xyz_host = std::make_unique<ColorSpaceXYZ>();
    scene = std::make_unique<Scene>(path);

    xyz_host->init();
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(c_material), scene->bsdfs,
                                         scene->num_bsdfs * sizeof(BSDF *)));
    CUDA_CHECK_RETURN(
        hipMemcpyToSymbol(HIP_SYMBOL(c_emitter), scene->emitters,
                           (scene->num_emitters + 1) * sizeof(Emitter *)));

    std::cout << "[RENDERER] Path tracer loaded: ";
    switch (scene->rdr_type) {
    case RendererType::MegaKernelPT: {
        rdr = std::make_unique<PathTracer<SingleTileScheduler>>(*scene);
        rdr->initialize_var_buffer();
        std::cout << "\tMegakernel Path Tracing (Static Scheduler).\n";
        break;
    }
    case RendererType::WavefrontPT: {
        rdr = std::make_unique<WavefrontPathTracer>(*scene);
        rdr->initialize_var_buffer();
        std::cout << "\tWavefront Path Tracing..\n";
        break;
    }
    case RendererType::MegaKernelLT: {
        rdr = std::make_unique<LightTracer>(
            *scene, scene->config.spec_constraint, scene->config.bidirectional,
            scene->config.caustic_scaling);
        if (scene->config.bidirectional)
            std::cout << "\tNaive Bidirectional Megakernel Light Tracing.\n";
        else
            std::cout << "\tMegakernel Light Tracing.\n";
        break;
    }
    case RendererType::MegaKernelVPT: {
        rdr = std::make_unique<VolumePathTracer>(*scene);
        std::cout << "\tVolumetric Path Tracer\n";
        break;
    }
    case RendererType::VoxelSDFPT: {
        std::cerr
            << "\tVoxelSDFPT is not implemented yet. Stay tuned. Rendering "
               "exits.\n";
        exit(0);
    }
    case RendererType::DepthTracing: {
        rdr = std::make_unique<DepthTracer>(*scene);
        std::cout << "\tDepth Tracing\n";
        break;
    }
    case RendererType::BVHCostViz: {
        rdr = std::make_unique<BVHCostVisualizer>(*scene);
        std::cout << "\tBVH Cost Visualizer\n";
        break;
    }
    case RendererType::MegaKernelPTDynamic: {
        rdr = std::make_unique<PathTracer<PreemptivePersistentTileScheduler>>(
            *scene);
        std::cout << "\tMegakernel Path Tracing (Dynamic Scheduler).\n";
        break;
    }
    default: {
        throw std::runtime_error("Unsupported renderer type.");
    }
    }
    scene->free_resources();
    rdr->set_seed_offset(seed_offset);
    rdr->update_camera(scene->cam);
    rdr->initialize_output_buffer();
}

void PythonRenderer::release() {
    xyz_host->destroy();
    valid = false;
}

void PythonRenderer::info() const { scene->print(); }
