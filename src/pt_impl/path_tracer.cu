#include "hip/hip_runtime.h"
// Copyright (C) 2025 Qianyue He
//
// This program is free software: you can redistribute it and/or
// modify it under the terms of the GNU Affero General Public License
// as published by the Free Software Foundation, either
// version 3 of the License, or (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See
// the GNU Affero General Public License for more details.
//
// You should have received a copy of the GNU Affero General
// Public License along with this program. If not, see
//
//             <https://www.gnu.org/licenses/>.

/**
 * @author Qianyue He
 * @brief Megakernel Path Tracer implementation
 * @date 2024.10.10
 */

#include "renderer/path_tracer.cuh"

static constexpr int SEED_SCALER =
    11467; // 11451 is not a prime, while 11467 is
static constexpr int SHFL_THREAD_X =
    5; // blockDim.x: 1 << SHFL_THREAD_X, by default, SHFL_THREAD_X is 4: 16
       // threads
static constexpr int SHFL_THREAD_Y =
    2; // blockDim.y: 1 << SHFL_THREAD_Y, by default, SHFL_THREAD_Y is 4: 16
       // threads

CPT_GPU_CONST Emitter *c_emitter[9];
CPT_GPU_CONST BSDF *c_material[48];

PathTracer::PathTracer(const Scene &scene, bool _verbose)
    : TracerBase(scene), verbose(_verbose), num_objs(scene.objects.size()),
      num_nodes(-1), num_emitter(scene.num_emitters),
      envmap_id(scene.envmap_id) {
    if (scene.bvh_available()) {
        size_t num_bvh = scene.obj_idxs.size();
        // Comment in case I forget: scene.nodes combines nodes_front and
        // nodes_back So the size of nodes is exactly twice the number of nodes
        num_nodes = scene.nodes.size() >> 1;
        num_cache = scene.cache_nodes.size();
        CUDA_CHECK_RETURN(hipMalloc(&_obj_idxs, num_bvh * sizeof(int)));
        CUDA_CHECK_RETURN(hipMalloc(&_nodes, 2 * num_nodes * sizeof(float4)));
        CUDA_CHECK_RETURN(
            hipMalloc(&_cached_nodes, num_cache * sizeof(uint4)));
        // note that BVH leaf node only stores the primitive to object mapping
        bvh_leaves =
            create_texture1d<int>(scene.obj_idxs.data(), num_bvh, _obj_idxs);
        nodes =
            create_texture1d<float4>(scene.nodes.data(), 2 * num_nodes, _nodes);
        CUDA_CHECK_RETURN(hipMemcpy(_cached_nodes, scene.cache_nodes.data(),
                                     sizeof(uint4) * num_cache,
                                     hipMemcpyHostToDevice));
    } else {
        throw std::runtime_error("BVH not available in scene. Abort.");
    }
    /**
     * Explanation: For envmap and point source, there is no attached object,
     * therefore, for scene that contains solely these emitters We can have a
     * zero emitter_prim_size, which is troublesome. We therefore needs to 'pad'
     * it, to at least the size of the first object, then the
     * sample_emitter_primitive can return values that will not cause any memory
     * leak
     */
    size_t emitter_prim_size =
               sizeof(int) * std::max(scene.emitter_prims.size(),
                                      (size_t)scene.objects.front().prim_num),
           actual_prim_size = sizeof(int) * scene.emitter_prims.size();
    CUDA_CHECK_RETURN(
        hipMallocManaged(&obj_info, num_objs * sizeof(CompactedObjInfo)));
    CUDA_CHECK_RETURN(hipMalloc(&camera, sizeof(DeviceCamera)));
    CUDA_CHECK_RETURN(hipMalloc(&emitter_prims, emitter_prim_size));
    CUDA_CHECK_RETURN(hipMemcpy(camera, &scene.cam, sizeof(DeviceCamera),
                                 hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(emitter_prims, scene.emitter_prims.data(),
                                 actual_prim_size, hipMemcpyHostToDevice));
    for (int i = 0; i < num_objs; i++)
        obj_info[i] = scene.objects[i].export_gpu();
}

PathTracer::~PathTracer() {
    CUDA_CHECK_RETURN(hipFree(obj_info));
    CUDA_CHECK_RETURN(hipFree(camera));
    CUDA_CHECK_RETURN(hipFree(emitter_prims));
    CUDA_CHECK_RETURN(hipDestroyTextureObject(bvh_leaves));
    CUDA_CHECK_RETURN(hipDestroyTextureObject(nodes));
    CUDA_CHECK_RETURN(hipFree(_obj_idxs));
    CUDA_CHECK_RETURN(hipFree(_nodes));
    CUDA_CHECK_RETURN(hipFree(_cached_nodes));
    if (verbose)
        printf("[Renderer] Path Tracer Object destroyed.\n");
}

CPT_CPU std::vector<uint8_t> PathTracer::render(const MaxDepthParams &md,
                                                int num_iter,
                                                bool gamma_correction) {
    printf("Rendering starts.\n");
    TicToc _timer("render_pt_kernel()", num_iter);
    size_t cached_size = std::max(num_cache * sizeof(uint4), sizeof(uint4));
    for (int i = 0; i < num_iter; i++) {
        // for more sophisticated renderer (like path tracer), shared_memory
        // should be used
        render_pt_kernel<false>
            <<<dim3(w >> SHFL_THREAD_X, h >> SHFL_THREAD_Y),
               dim3(1 << SHFL_THREAD_X, 1 << SHFL_THREAD_Y), cached_size>>>(
                *camera, verts, norms, uvs, obj_info, emitter_prims, bvh_leaves,
                nodes, _cached_nodes, image, md, output_buffer, nullptr,
                num_emitter, i * SEED_SCALER + seed_offset, num_nodes,
                accum_cnt, num_cache, envmap_id);
        CUDA_CHECK_RETURN(hipDeviceSynchronize());
        printProgress(i, num_iter);
    }
    printf("\n");
    return image.export_cpu(1.f / num_iter, gamma_correction);
}

CPT_CPU void PathTracer::render_online(const MaxDepthParams &md,
                                       bool gamma_corr) {
    CUDA_CHECK_RETURN(hipGraphicsMapResources(1, &pbo_resc, 0));
    size_t _num_bytes = 0,
           cached_size = std::max(num_cache * sizeof(uint4), sizeof(uint4));
    // if we have an illegal memory access here: check whether you have a valid
    // emitter in the xml scene description file. it might be possible that
    // having no valid emitter triggers an illegal memory access
    CUDA_CHECK_RETURN(hipGraphicsResourceGetMappedPointer(
        (void **)&output_buffer, &_num_bytes, pbo_resc));
    accum_cnt++;
    render_pt_kernel<true>
        <<<dim3(w >> SHFL_THREAD_X, h >> SHFL_THREAD_Y),
           dim3(1 << SHFL_THREAD_X, 1 << SHFL_THREAD_Y), cached_size>>>(
            *camera, verts, norms, uvs, obj_info, emitter_prims, bvh_leaves,
            nodes, _cached_nodes, image, md, output_buffer, nullptr,
            num_emitter, accum_cnt * SEED_SCALER + seed_offset, num_nodes,
            accum_cnt, num_cache, envmap_id, gamma_corr);
    CUDA_CHECK_RETURN(hipGraphicsUnmapResources(1, &pbo_resc, 0));
}

CPT_CPU const float *PathTracer::render_raw(const MaxDepthParams &md,
                                            bool gamma_corr) {
    size_t cached_size = std::max(num_cache * sizeof(uint4), sizeof(uint4));
    accum_cnt++;
    render_pt_kernel<true>
        <<<dim3(w >> SHFL_THREAD_X, h >> SHFL_THREAD_Y),
           dim3(1 << SHFL_THREAD_X, 1 << SHFL_THREAD_Y), cached_size>>>(
            *camera, verts, norms, uvs, obj_info, emitter_prims, bvh_leaves,
            nodes, _cached_nodes, image, md, output_buffer, var_buffer,
            num_emitter, accum_cnt * SEED_SCALER + seed_offset, num_nodes,
            accum_cnt, num_cache, envmap_id, gamma_corr);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    return output_buffer;
}
