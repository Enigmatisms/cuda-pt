#include "hip/hip_runtime.h"
// Copyright (C) 2025 Qianyue He
//
// This program is free software: you can redistribute it and/or
// modify it under the terms of the GNU Affero General Public License
// as published by the Free Software Foundation, either
// version 3 of the License, or (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See
// the GNU Affero General Public License for more details.
//
// You should have received a copy of the GNU Affero General
// Public License along with this program. If not, see
//
//             <https://www.gnu.org/licenses/>.

/**
 * @author Qianyue He
 * @brief Megakernel Path Tracer implementation
 * @date 2024.10.10
 */

#include "renderer/light_tracer.cuh"

static constexpr int SEED_SCALER = 11467; //-4!
static constexpr int SHFL_THREAD_X =
    4; // blockDim.x: 1 << SHFL_THREAD_X, by default, SHFL_THREAD_X is 4: 16
       // threads
static constexpr int SHFL_THREAD_Y =
    3; // blockDim.y: 1 << SHFL_THREAD_Y, by default, SHFL_THREAD_Y is 4: 16
       // threads

CPT_CPU std::vector<uint8_t> LightTracer::render(const MaxDepthParams &md,
                                                 int num_iter,
                                                 bool gamma_correction) {
    printf("Rendering starts.\n");
    TicToc _timer("render_lt_kernel()", num_iter);
    size_t cached_size = num_cache * sizeof(uint4);
    for (int i = 0; i < num_iter; i++) {
        // for more sophisticated renderer (like path tracer), shared_memory
        // should be used
        if (bidirectional) {
            render_pt_kernel<SingleTileScheduler, true>
                <<<dim3(w >> SHFL_THREAD_X, h >> SHFL_THREAD_Y),
                   dim3(1 << SHFL_THREAD_X, 1 << SHFL_THREAD_Y), cached_size>>>(
                    *camera, verts, norms, uvs, obj_info, emitter_prims,
                    bvh_leaves, nodes, _cached_nodes, image, md, nullptr,
                    nullptr, num_emitter, accum_cnt * SEED_SCALER + seed_offset,
                    num_nodes, accum_cnt);
            CUDA_CHECK_RETURN(hipDeviceSynchronize());
        }
        render_lt_kernel<false>
            <<<dim3(w >> SHFL_THREAD_X, h >> SHFL_THREAD_Y),
               dim3(1 << SHFL_THREAD_X, 1 << SHFL_THREAD_Y), cached_size>>>(
                *camera, verts, norms, uvs, obj_info, emitter_prims, bvh_leaves,
                nodes, _cached_nodes, image, md, nullptr, num_emitter,
                i * SEED_SCALER + seed_offset, num_nodes, spec_constraint);
        CUDA_CHECK_RETURN(hipDeviceSynchronize());
        printProgress(i, num_iter);
    }
    printf("\n");
    return image.export_cpu(1.f / num_iter, gamma_correction, true);
}

CPT_CPU void LightTracer::render_online(const MaxDepthParams &md,
                                        bool gamma_corr) {
    CUDA_CHECK_RETURN(hipGraphicsMapResources(1, &pbo_resc, 0));
    size_t _num_bytes = 0, cached_size = num_cache * sizeof(uint4);
    CUDA_CHECK_RETURN(hipGraphicsResourceGetMappedPointer(
        (void **)&output_buffer, &_num_bytes, pbo_resc));

    accum_cnt++;
    if (bidirectional) {
        render_pt_kernel<SingleTileScheduler, false>
            <<<dim3(w >> SHFL_THREAD_X, h >> SHFL_THREAD_Y),
               dim3(1 << SHFL_THREAD_X, 1 << SHFL_THREAD_Y), cached_size>>>(
                *camera, verts, norms, uvs, obj_info, emitter_prims, bvh_leaves,
                nodes, _cached_nodes, image, md, output_buffer, nullptr,
                num_emitter, accum_cnt * SEED_SCALER + seed_offset, num_nodes,
                accum_cnt, num_cache, false);
        CUDA_CHECK_RETURN(hipDeviceSynchronize());
    }
    render_lt_kernel<true>
        <<<dim3(w >> SHFL_THREAD_X, h >> SHFL_THREAD_Y),
           dim3(1 << SHFL_THREAD_X, 1 << SHFL_THREAD_Y), cached_size>>>(
            *camera, verts, norms, uvs, obj_info, emitter_prims, bvh_leaves,
            nodes, _cached_nodes, image, md, output_buffer, num_emitter,
            accum_cnt * SEED_SCALER + seed_offset, num_nodes, accum_cnt,
            num_cache, spec_constraint, caustic_scaling, gamma_corr);
    CUDA_CHECK_RETURN(hipGraphicsUnmapResources(1, &pbo_resc, 0));
}

CPT_CPU const float *LightTracer::render_raw(const MaxDepthParams &md,
                                             bool gamma_corr) {
    size_t cached_size = std::max(num_cache * sizeof(uint4), sizeof(uint4));
    accum_cnt++;
    if (bidirectional) {
        render_pt_kernel<SingleTileScheduler, false>
            <<<dim3(w >> SHFL_THREAD_X, h >> SHFL_THREAD_Y),
               dim3(1 << SHFL_THREAD_X, 1 << SHFL_THREAD_Y), cached_size>>>(
                *camera, verts, norms, uvs, obj_info, emitter_prims, bvh_leaves,
                nodes, _cached_nodes, image, md, output_buffer, nullptr,
                num_emitter, accum_cnt * SEED_SCALER + seed_offset, num_nodes,
                accum_cnt, num_cache, false);
        CUDA_CHECK_RETURN(hipDeviceSynchronize());
    }
    render_lt_kernel<true>
        <<<dim3(w >> SHFL_THREAD_X, h >> SHFL_THREAD_Y),
           dim3(1 << SHFL_THREAD_X, 1 << SHFL_THREAD_Y), cached_size>>>(
            *camera, verts, norms, uvs, obj_info, emitter_prims, bvh_leaves,
            nodes, _cached_nodes, image, md, output_buffer, num_emitter,
            accum_cnt * SEED_SCALER + seed_offset, num_nodes, accum_cnt,
            num_cache, spec_constraint, caustic_scaling, gamma_corr);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    return output_buffer;
}
