#include "hip/hip_runtime.h"
// Copyright (C) 2025 Qianyue He
//
// This program is free software: you can redistribute it and/or
// modify it under the terms of the GNU Affero General Public License
// as published by the Free Software Foundation, either
// version 3 of the License, or (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See
// the GNU Affero General Public License for more details.
//
// You should have received a copy of the GNU Affero General
// Public License along with this program. If not, see
//
//             <https://www.gnu.org/licenses/>.

/**
 * @author Qianyue He
 * @brief Spatial BVH construction main logic
 * @date 2025.5.25
 */

#include "core/bvh_opt.cuh"
#include "core/proc_geometry.cuh"
#include "core/stats.h"
#include <algorithm>
#include <cassert>
#include <cinttypes>
#include <numeric>
#include <optional>

#include <atomic>
#include <atomic_queue/atomic_queue.h>
#include <condition_variable>
#include <future>
#include <mutex>
#include <thread>

static constexpr int num_bins = 64;
static constexpr int num_sbins = 128; // spatial bins
static constexpr int no_div_threshold = 2;
static constexpr int sah_split_threshold = 8;
// A cluster with all the primitive centroid within a small range [less than
// 1e-3] is ill-posed. If there is more than 64 primitives, the primitives will
// be discarded
static constexpr bool SSP_DEBUG = true;
static constexpr float traverse_cost = 0.2f;
static constexpr float spatial_traverse_cost = 0.21f;
static constexpr int max_allowed_depth = 96;
// when number of triangles to process is less than the following,
// the task will be executed locally instead of being queued
static constexpr int queue_workload_threshold = 512;
// when number of triangles to process is greater than the following,
// `update_bin` will employ thread pool to accelerate binning
static constexpr int workload_threshold = 128;
#ifdef OPENMP_ENABLED
#define OMP_GET_THREAD_ID omp_get_thread_num()
static constexpr int number_of_workers = 8;
#else
#define OMP_GET_THREAD_ID 0
static constexpr int number_of_workers = 1;
#endif // OPENMP_ENABLED
static int max_depth = 0;

using SBVHBuilderTaskKey = uint64_t;
// Wrapping building of a SBVH Node as a Task for queuing
struct SBVHBuilderTask {
    SBVHNode *cur_node;
    int depth;

    std::array<SBVHBuilderTask, 2> get_child_tasks() const {
        return {SBVHBuilderTask{cur_node->lchild, depth + 1},
                SBVHBuilderTask{cur_node->rchild, depth + 1}};
    }
    bool is_leaf() const { return !cur_node->non_leaf(); }

    SBVHBuilderTaskKey get_key() const {
        static_assert(max_allowed_depth < 256);
        static_assert(std::is_same_v<uint64_t, SBVHBuilderTaskKey>);
        static_assert(sizeof(SBVHNode *) <= 8);
        return uint64_t(uintptr_t(cur_node)) | uint64_t(depth) << 56ULL;
    }
    static SBVHBuilderTask from_key(SBVHBuilderTaskKey key) {
        return {.cur_node = (SBVHNode *)uintptr_t(key & 0x00FFFFFFFFFFFFFFULL),
                .depth = int(key >> 56ULL)};
    }
};

// A simple thread pool with 1 thread and 1 task
class SBVHBuilderThread {
  private:
    mutable std::optional<std::function<void()>> task;
    mutable std::condition_variable cv;
    mutable std::mutex mutex;
    mutable std::thread thread;
    mutable bool run_flag;

    void worker_func() {
        for (std::function<void()> cur_task;;) {
            {
                std::unique_lock lock{mutex};
                cv.wait(lock, [this] { return task.has_value() || !run_flag; });
                if (!run_flag && !task.has_value())
                    return;
                cur_task = std::move(task.value());
                task.reset();
            }
            cur_task();
        }
    }

  public:
    SBVHBuilderThread() : run_flag{true} {
        thread = std::thread(&SBVHBuilderThread::worker_func, this);
    }
    ~SBVHBuilderThread() {
        {
            std::scoped_lock lock{mutex};
            run_flag = false;
        }
        cv.notify_one();
        thread.join();
    }
    template <typename Func_T, typename... Arg_Ts,
              typename Result_T = std::invoke_result_t<Func_T, Arg_Ts...>>
    std::future<Result_T> push(Func_T &&func, Arg_Ts &&...args) const {
        auto push_task =
            std::make_shared<std::packaged_task<Result_T()>>(std::bind(
                std::forward<Func_T>(func), std::forward<Arg_Ts>(args)...));
        std::future<Result_T> future = push_task->get_future();
        {
            std::scoped_lock lock{mutex};
            task = [push_task] { (*push_task)(); };
        }
        cv.notify_one();
        return future;
    }
};

struct SBVHBuilderThreadID {
    // `local` means the index in a SBVHBuilderThreadSpan
    int global, local;
};

// A span of threads that can be used for task execution
class SBVHBuilderThreadSpan {
  private:
    // parallel threads indexed as 1, 2, ..., parallel_threads.size(),
    // the main thread is indexed as 0.
    // (parallel_threads.size() + 1) worker threads totally
    const std::vector<SBVHBuilderThread> &parallel_threads;

    // defining the span of threads in range 0, 1, ..., parallel_threads.size()
    int thread_base, thread_count;

  public:
    explicit SBVHBuilderThreadSpan(
        const std::vector<SBVHBuilderThread> &parallel_threads)
        : parallel_threads{parallel_threads}, thread_base{0},
          thread_count((int)parallel_threads.size() + 1) {}
    SBVHBuilderThreadSpan(
        const std::vector<SBVHBuilderThread> &parallel_threads, int thread_base,
        int thread_count)
        : parallel_threads{parallel_threads}, thread_base{thread_base},
          thread_count{thread_count} {}

    std::array<SBVHBuilderThreadSpan, 2>
    get_child_spans(const SBVHBuilderTask &lchild_task,
                    const SBVHBuilderTask &rchild_task) const {
        // distribute threads in proportion to prim count
        int lchild_prim_cnt = lchild_task.cur_node->prims.size();
        int rchild_prim_cnt = rchild_task.cur_node->prims.size();

        int lchild_thread_cnt = std::clamp(
            int(std::round(float(lchild_prim_cnt * thread_count) /
                           float(lchild_prim_cnt + rchild_prim_cnt))),
            0, thread_count);
        int rchild_thread_cnt = thread_count - lchild_thread_cnt;
        return {SBVHBuilderThreadSpan{parallel_threads, thread_base,
                                      lchild_thread_cnt},
                SBVHBuilderThreadSpan{parallel_threads,
                                      thread_base + lchild_thread_cnt,
                                      rchild_thread_cnt}};
    }

    bool should_queued() const { return thread_count == 0; }
    bool can_parallelize() const { return thread_count > 1; }
    int get_parallelism() const { return thread_count; }
    SBVHBuilderThreadID get_thread_id(int thd_ofst = 0) const {
        return {.global = thread_base + thd_ofst, .local = thd_ofst};
    }

    // run_... functions must be called on the first thread of a span
    template <typename Mapper_T, typename Reducer_T>
    void run_parallel_for(int size, Mapper_T &&mapper,
                          Reducer_T &&reducer) const {
        // assert can_parallelize()

        int block_size = size / thread_count;
        const auto func = [this, block_size, size,
                           &mapper](SBVHBuilderThreadID thd_id) {
            int begin = block_size * thd_id.local;
            int end =
                thd_id.local == thread_count - 1 ? size : begin + block_size;
            for (int i = begin; i < end; ++i)
                mapper(thd_id, i);
        };

        std::vector<std::future<void>> futures(thread_count - 1);
        for (int thd_ofst = 1; thd_ofst < thread_count; ++thd_ofst) {
            SBVHBuilderThreadID thd_id = get_thread_id(thd_ofst);
            futures[thd_id.local - 1] =
                parallel_threads[thd_id.global - 1].push(func, thd_id);
        }
        func(get_thread_id());
        reducer(get_thread_id());
        for (int thd_ofst = 1; thd_ofst < thread_count; ++thd_ofst) {
            SBVHBuilderThreadID thd_id = get_thread_id(thd_ofst);
            futures[thd_id.local - 1].wait();
            reducer(thd_id);
        }
    }
    template <typename Func_T, typename Result_T = std::invoke_result_t<Func_T>>
    std::future<Result_T> run_async(Func_T &&func) const {
        // assert(get_thread_id().global != 0)
        // must not be called on the global first thread
        return parallel_threads[get_thread_id().global - 1].push(func);
    }
};

SplitAxis SBVHNode::max_extent_axis(const std::vector<BVHInfo> &bvhs,
                                    float &min_r, float &interval) const {
    Vec3 min_ctr = Vec3(AABB_INVALID_DIST), max_ctr = Vec3(-AABB_INVALID_DIST);

    // Note: SBVH requires that the AABB of the child node to be <= to the
    // father's AABB. Therefore, if there is a spatial split followed by an
    // object split, some parts of the primitives might be outside of the AABB
    // of the father node. As a result, the centroids, and even the bins might
    // be outside of the AABB of the father node. So, for max extent and object
    // spatial binning, we need to clip the range inside the AABB of the father
    // node . This is different from BVH nodes (since the AABBs of their child
    // nodes must reside within the AABBs of the father nodes)
    for (int bvh_id : prims) {
        Vec3 ctr = bound.clamp(bvhs[bvh_id].centroid);
        min_ctr.minimized(ctr);
        max_ctr.maximized(ctr);
    }

    Vec3 diff = max_ctr - min_ctr;
    float max_diff = diff.x();
    min_r = min_ctr[0] - AABB_EPS;
    int split_axis = 0;
    if (diff.y() > max_diff) {
        max_diff = diff.y();
        split_axis = 1;
        min_r = min_ctr[1] - AABB_EPS;
    }
    if (diff.z() > max_diff) {
        max_diff = diff.z();
        split_axis = 2;
        min_r = min_ctr[2] - AABB_EPS;
    }
    if (diff.max_elem() < 1e-3) {
        return SplitAxis::AXIS_NONE;
    }
    interval = (max_diff + AABB_EPS * 2.f) / float(num_bins);
    return SplitAxis(split_axis);
}

template <int N> void SpatialSplitter<N>::bound_all_bins() {
    for (int i = 0; i < N; i++) {
        bounds[i] ^= bound;
    }
}

template <int N>
bool SpatialSplitter<N>::update_triangle(
    std::vector<Vec3> &&points, std::array<AABB, N> &_bounds,
    std::array<std::vector<int>, N> &_enter_tris,
    std::array<std::vector<int>, N> &_exit_tris,
    std::vector<AABB> &_clip_poly_aabbs, int prim_id) const {
    // Note that spatial split triangles can have parts outside of the AABB
    // so we must not assume that AABB is tight (object-ly, but spatially)

    int min_axis_v = N, max_axis_v = -1;

    std::vector<Vec3> clipped_poly =
        aabb_triangle_clipping(bound, std::move(points));

    if (clipped_poly.empty()) {
        if constexpr (SSP_DEBUG) {
            std::cerr << "[SBVH Warn] Primitive " << prim_id
                      << " discarded due to being degenerated after triangle "
                         "clipping. This should not happen.\n";
        }
        return false;
    }

    AABB clip_aabb(AABB_INVALID_DIST, -AABB_INVALID_DIST, 0, 0);
    Vec3 sp = clipped_poly.back();
    for (size_t i = 0; i < clipped_poly.size(); i++) {
        Vec3 ep = clipped_poly[i], old_ep = ep;
        if (employ_unsplit)
            clip_aabb.extend(ep);

        if (sp[axis] > ep[axis])
            std::swap(sp, ep);

        Vec3 dir = ep - sp;
        float dim_v = dir[axis];
        int s_idx = get_bin_id(sp), e_idx = get_bin_id(ep);
        min_axis_v = std::min(min_axis_v, s_idx);
        max_axis_v = std::max(max_axis_v, e_idx);

        if (std::abs(dim_v) < 1e-5f) {
            _bounds[s_idx].extend(sp);
            _bounds[e_idx].extend(ep);
        } else {
            dir *= 1.f / dim_v;
            float d2bin_start =
                s_pos + interval * static_cast<float>(s_idx) - sp[axis];
            Vec3 pt = sp.advance(dir, d2bin_start);
            for (int id = s_idx; id <= e_idx; id++) {
                AABB &aabb = _bounds[id];
                aabb.extend(bound.clamp(s_idx == id ? sp : pt));
                pt = pt.advance(dir, interval);
                aabb.extend(bound.clamp(e_idx == id ? ep : pt));
            }
        }
        sp = std::move(old_ep);
    }

    if (employ_unsplit) {
        auto packed_indices = reinterpret_cast<int16_t *>(&clip_aabb.__bytes1);
        packed_indices[0] = min_axis_v;
        packed_indices[1] = max_axis_v;
        clip_aabb.__bytes2 = prim_id;
        _clip_poly_aabbs.emplace_back(std::move(clip_aabb));
    }
    _enter_tris[min_axis_v].push_back(prim_id);
    _exit_tris[max_axis_v].push_back(prim_id);
    return true;
}

// declared for parallel processing
struct ChoppedBinningData {
    std::array<AABB, num_sbins> bounds;
    std::array<std::vector<int>, num_sbins> enter_tris;
    std::array<std::vector<int>, num_sbins> exit_tris;
    std::vector<AABB> clip_poly_aabbs;

    ChoppedBinningData() {
        for (int i = 0; i < num_sbins; i++) {
            bounds[i].clear();
        }
    }
};

template <int N>
void SpatialSplitter<N>::update_bins(const std::vector<Vec3> &points1,
                                     const std::vector<Vec3> &points2,
                                     const std::vector<Vec3> &points3,
                                     /* possibly, add sphere flag later */
                                     const SBVHBuilderThreadSpan &threads,
                                     const SBVHNode *const cur_node) {
    // the following can be made faster by partitioning and multi-threading
    if (threads.can_parallelize() && cur_node->size() >= workload_threshold) {
        // multi-thread implementation
        std::vector<ChoppedBinningData> all_data(threads.get_parallelism());

        ChoppedBinningData result;
        size_t clip_aabb_size = 0;

        threads.run_parallel_for(
            cur_node->size(),
            [&](SBVHBuilderThreadID thread_id, int i) {
                int prim_id = cur_node->prims[i];
                ChoppedBinningData &local_data = all_data[thread_id.local];
                update_triangle(
                    {points1[prim_id], points2[prim_id], points3[prim_id]},
                    local_data.bounds, local_data.enter_tris,
                    local_data.exit_tris, local_data.clip_poly_aabbs, prim_id);
            },
            [&](SBVHBuilderThreadID thread_id) {
                auto &local_data = all_data[thread_id.local];
                for (int bin_id = 0; bin_id < N; bin_id++) {
                    result.bounds[bin_id] += local_data.bounds[bin_id];
                    result.enter_tris[bin_id].insert(
                        result.enter_tris[bin_id].end(),
                        local_data.enter_tris[bin_id].begin(),
                        local_data.enter_tris[bin_id].end());
                    result.exit_tris[bin_id].insert(
                        result.exit_tris[bin_id].end(),
                        local_data.exit_tris[bin_id].begin(),
                        local_data.exit_tris[bin_id].end());
                }
                if (employ_unsplit) {
                    clip_aabb_size += local_data.clip_poly_aabbs.size();
                }
            });

        if (clip_aabb_size > 0) {
            result.clip_poly_aabbs.reserve(clip_aabb_size);
            for (ChoppedBinningData &local_data : all_data) {
                result.clip_poly_aabbs.insert(
                    result.clip_poly_aabbs.end(),
                    local_data.clip_poly_aabbs.begin(),
                    local_data.clip_poly_aabbs.end());
            }
        }
        bounds = std::move(result.bounds);
        enter_tris = std::move(result.enter_tris);
        exit_tris = std::move(result.exit_tris);
        clip_poly_aabbs = std::move(result.clip_poly_aabbs);
    } else {
        // single-threaded implement
        for (int prim_id : cur_node->prims) {
            update_triangle(
                {points1[prim_id], points2[prim_id], points3[prim_id]}, bounds,
                enter_tris, exit_tris, clip_poly_aabbs, prim_id);
        }
    }
}

template <int N>
float SpatialSplitter<N>::eval_spatial_split(int &seg_bin_idx,
                                             int node_prim_cnt,
                                             float trav_cost) {
    float min_cost = 5e9f;

    std::array<float, N> fwd_areas, bwd_areas;
    fwd_areas.fill(0);
    bwd_areas.fill(0);

    AABB fwd_bound(AABB_INVALID_DIST, -AABB_INVALID_DIST, 0, 0),
        bwd_bound(AABB_INVALID_DIST, -AABB_INVALID_DIST, 0, 0);
    for (int i = 0; i < N; i++) {
        fwd_bound += bounds[i];
        fwd_areas[i] = fwd_bound.area();
        lprim_cnts[i] = enter_tris[i].size();
        if (i > 0) {
            lprim_cnts[i] += lprim_cnts[i - 1];
            bwd_bound += bounds[N - i];
            bwd_areas[N - 1 - i] = bwd_bound.area();
            // the same as BVH, the [N-1] will be 0, since seg_idx can never be
            // N - 1, also, exit_tris[0] will not be accessed (since
            // unnecessary)
            rprim_cnts[N - 1 - i] = exit_tris[N - i].size() + rprim_cnts[N - i];
        }
    }
    float node_inv_area = 1.f / bound.area();

    for (int i = 0; i < N - 1; i++) {
        float cost =
            trav_cost + node_inv_area * (float(lprim_cnts[i]) * fwd_areas[i] +
                                         float(rprim_cnts[i]) * bwd_areas[i]);
        if (cost < min_cost) {
            min_cost = cost;
            seg_bin_idx = i;
        }
    }
    return min_cost;
}

template <int N>
std::pair<AABB, AABB>
SpatialSplitter<N>::apply_unsplit_reference(std::vector<int> &left_prims,
                                            std::vector<int> &right_prims,
                                            float &min_cost, int seg_bin_idx) {
    // the min_cost is not a standard SAH cost. min_cost(here) = (min_cost -
    // traverse_cost) / node_inv_area;
    int lchild_cnt = lprim_cnts[seg_bin_idx],
        rchild_cnt = rprim_cnts[seg_bin_idx];
    // for child node with only one primitive, no need for unsplitting

    AABB fwd_bound(AABB_INVALID_DIST, -AABB_INVALID_DIST, 0, 0),
        bwd_bound(AABB_INVALID_DIST, -AABB_INVALID_DIST, 0, 0);
    for (int i = 0; i <= seg_bin_idx; i++) // calculate child node bound
        fwd_bound += bounds[i];
    for (int i = seg_bin_idx + 1; i < N; i++)
        bwd_bound += bounds[i];
    for (size_t i = 0; i < clip_poly_aabbs.size(); i++) {
        if (lchild_cnt <= 1 || rchild_cnt <= 1)
            break;
        const AABB &aabb = clip_poly_aabbs[i];
        // if not a straddled reference, skip
        auto packed_indices = reinterpret_cast<const int16_t *>(&aabb.__bytes1);
        if (packed_indices[0] > seg_bin_idx || packed_indices[1] <= seg_bin_idx)
            continue;
        float to_left_cost = (fwd_bound + aabb).area() * lchild_cnt +
                             bwd_bound.area() * (rchild_cnt - 1),
              to_right_cost = fwd_bound.area() * (lchild_cnt - 1) +
                              (aabb + bwd_bound).area() * rchild_cnt;
        if (to_left_cost >= min_cost && to_right_cost >= min_cost)
            continue;
        int index = aabb.__bytes2;
        if (to_left_cost < to_right_cost) { // the less one must < min_cost
            fwd_bound += aabb;
            min_cost = to_left_cost;
            unsplit_right.emplace(index);
            rchild_cnt--;
        } else {
            bwd_bound += aabb;
            min_cost = to_right_cost;
            unsplit_left.emplace(index);
            lchild_cnt--;
        }
    }
#define FILTER_EMPLACE(src, dst, filter, cnt, begin_i, end_i)                  \
    dst.reserve(cnt);                                                          \
    for (int i = begin_i; i <= end_i; i++) {                                   \
        const auto &idxs = src[i];                                             \
        for (int prim_idx : idxs) {                                            \
            if (filter.count(prim_idx))                                        \
                continue;                                                      \
            dst.push_back(prim_idx);                                           \
        }                                                                      \
    }

    FILTER_EMPLACE(enter_tris, left_prims, unsplit_left, lchild_cnt, 0,
                   seg_bin_idx)
    FILTER_EMPLACE(exit_tris, right_prims, unsplit_right, rchild_cnt,
                   seg_bin_idx + 1, N - 1)
#undef FILTER_EMPLACE
    return std::make_pair(fwd_bound, bwd_bound);
}

template <int N>
std::pair<AABB, AABB>
SpatialSplitter<N>::apply_spatial_split(std::vector<int> &left_prims,
                                        std::vector<int> &right_prims,
                                        int seg_bin_idx) {
    left_prims.reserve(lprim_cnts[seg_bin_idx]);
    for (int i = 0; i <= seg_bin_idx; i++) {
        left_prims.insert(left_prims.end(), enter_tris[i].begin(),
                          enter_tris[i].end());
    }
    right_prims.reserve(rprim_cnts[seg_bin_idx]);
    for (int i = seg_bin_idx + 1; i < N; i++) {
        right_prims.insert(right_prims.end(), exit_tris[i].begin(),
                           exit_tris[i].end());
    }

    if constexpr (SSP_DEBUG) {
        if (left_prims.empty() || right_prims.empty()) {
            std::cerr << "Spatial split results in empty child nodes: "
                      << left_prims.size() << ", " << right_prims.size()
                      << std::endl;
            throw std::runtime_error("Spatial split failed.");
        }
    }

    AABB fwd_bound(AABB_INVALID_DIST, -AABB_INVALID_DIST, 0, 0),
        bwd_bound(AABB_INVALID_DIST, -AABB_INVALID_DIST, 0, 0);
    for (int i = 0; i <= seg_bin_idx; i++) // calculate child node bound
        fwd_bound += bounds[i];
    for (int i = seg_bin_idx + 1; i < N; i++)
        bwd_bound += bounds[i];
    return std::make_pair(fwd_bound, bwd_bound);
}

bool spatial_split_criteria(float root_area, float intrs_area, int num_prims) {
    // SS can be applied if overlap relative to root >= the following. This
    // factor is in fact mentioned in the original paper.
    static constexpr float root_overlap_factor = 1e-5f;

    return intrs_area > root_overlap_factor * root_area;
}

// TODO(heqianyue): note that we currently don't support
// sphere primitive. Support it would be straightforward:
// overload the 'update' function for spheres
void node_sbvh_SAH(const std::vector<Vec3> &points1,
                   const std::vector<Vec3> &points2,
                   const std::vector<Vec3> &points3,
                   const std::vector<BVHInfo> &bvh_infos,
                   const SBVHBuilderThreadSpan &threads,
                   const SBVHBuilderTask &task, float root_area,
                   int max_prim_node = 16, bool ref_unsplit = true) {

    auto cur_node = task.cur_node;
    auto depth = task.depth;

    auto process_leaf = [&]() {
        // leaf node processing function
        cur_node->axis = AXIS_NONE;
        cur_node->prim_num() = static_cast<int>(cur_node->size());
    };
    float min_range = 0, interval = 0;
    // Step 1: decide the axis that expands the maximum extent of space
    SplitAxis max_axis =
        cur_node->max_extent_axis(bvh_infos, min_range, interval);

    if (cur_node->size() <= no_div_threshold || depth >= max_allowed_depth ||
        max_axis == SplitAxis::AXIS_NONE) {
        // if the node is small, or father nodes and child nodes share the same
        // size (spatial split duplication) for too many times, we'll create a
        // leaf node
        return process_leaf();
    }
    AABB fwd_bound(1e5f, -1e5f, 0, 0), bwd_bound(1e5f, -1e5f, 0, 0);
    const int prim_num = cur_node->size();
    float min_cost = 5e9f, node_prim_cnt = float(prim_num);

    std::vector<int> lchild_idxs, rchild_idxs;
    if (prim_num > sah_split_threshold) { // SAH
        // Step 2: binning the space
        std::array<AxisBins, num_bins> idx_bins;
        for (int bvh_id : cur_node->prims) {
            // some of the primitives might just have their centroids outside of
            // all the bins (as a result from spatial split followed by an
            // object split)
            int index = std::clamp(
                static_cast<int>(
                    (bvh_infos[bvh_id].centroid[max_axis] - min_range) /
                    interval),
                0, num_bins - 1);
            idx_bins[index].push(bvh_infos[bvh_id]);
        }
        for (int i = 0; i < num_bins; i++) {
            idx_bins[i].bound ^= cur_node->bound;
        }

        // Step 3: forward-backward linear sweep for heuristic calculation
        std::array<int, num_bins> prim_cnts;
        std::array<float, num_bins> fwd_areas, bwd_areas;

        prim_cnts.fill(0);
        fwd_areas.fill(0);
        bwd_areas.fill(0);
        for (int i = 0; i < num_bins; i++) {
            fwd_bound += idx_bins[i].bound;
            prim_cnts[i] = idx_bins[i].prim_cnt;
            fwd_areas[i] = fwd_bound.area();
            if (i > 0) {
                bwd_bound += idx_bins[num_bins - i].bound;
                bwd_areas[num_bins - 1 - i] = bwd_bound.area();
            }
        }

        float node_inv_area = 1. / fwd_bound.area();
        std::partial_sum(prim_cnts.begin(), prim_cnts.end(), prim_cnts.begin());

        // Step 4: use the calculated area to computed the segment boundary, for
        // SBVH there is no need using spatial overlap penalty for BVH
        int seg_bin_idx = 0;
        for (int i = 0; i < num_bins - 1; i++) {
            float cost =
                traverse_cost +
                node_inv_area *
                    (float(prim_cnts[i]) * fwd_areas[i] +
                     (node_prim_cnt - float(prim_cnts[i])) * bwd_areas[i]);
            if (cost < min_cost) {
                min_cost = cost;
                seg_bin_idx = i;
            }
        }

        fwd_bound.clear();
        bwd_bound.clear();
        for (int i = 0; i <= seg_bin_idx; i++) // calculate child node bound
            fwd_bound += idx_bins[i].bound;
        for (int i = seg_bin_idx + 1; i < num_bins; i++)
            bwd_bound += idx_bins[i].bound;

        bool spatial_split_applied = false;
        if (spatial_split_criteria(
                root_area, fwd_bound.intersection_area(bwd_bound), prim_num)) {
            SpatialSplitter<num_sbins> ssp(cur_node->bound, max_axis,
                                           ref_unsplit);
            ssp.update_bins(points1, points2, points3, threads, cur_node);

            int sbvh_seg_idx = 0;
            float sbvh_cost = ssp.eval_spatial_split(
                sbvh_seg_idx, cur_node->size(), spatial_traverse_cost);
            // printf("SBVH: spatial split cost: %f, object split cost: %f\n",
            // sbvh_cost, min_cost);
            if (sbvh_cost < min_cost &&
                sbvh_cost < node_prim_cnt) { // Spatial split, actually node num
                                             // is not capped here
                max_axis = ssp.get_axis();
                if (ssp.employ_ref_unsplit()) {
                    sbvh_cost = (sbvh_cost - spatial_traverse_cost) *
                                cur_node->bound.area();
                    float old_sbvh_cost = sbvh_cost;
                    std::tie(fwd_bound, bwd_bound) =
                        ssp.apply_unsplit_reference(lchild_idxs, rchild_idxs,
                                                    sbvh_cost, sbvh_seg_idx);
                    if (old_sbvh_cost > sbvh_cost + THP_EPS) {
                        reinterpret_cast<int &>(max_axis) |=
                            SplitAxis::REF_UNSPLIT;
                    }
                } else {
                    std::tie(fwd_bound, bwd_bound) = ssp.apply_spatial_split(
                        lchild_idxs, rchild_idxs, sbvh_seg_idx);
                }
                fwd_bound.grow(1e-5f);
                bwd_bound.grow(1e-5f);
                spatial_split_applied = true;
            }
        }

        if (!spatial_split_applied &&
            (min_cost < node_prim_cnt ||
             prim_num > max_prim_node)) { // object split
            // 1. SBVH is not applied ; 2. when the cost of splitting is lower
            // or 3. when there are more primitives than allowed We cannot
            // partition here, since partition will change the index of the BVH
            lchild_idxs.reserve(prim_num / 2);
            rchild_idxs.reserve(prim_num / 2);
            float pivot = min_range + interval * float(seg_bin_idx + 1);
            for (int bvh_id : cur_node->prims) {
                const BVHInfo &bvh = bvh_infos[bvh_id];
                if (bvh.centroid[max_axis] < pivot) {
                    lchild_idxs.push_back(bvh_id);
                } else {
                    rchild_idxs.push_back(bvh_id);
                }
            }
        }
    } else { // equal primitive number split (two nodes have identical
             // primitives)
        std::vector<std::pair<float, int>> valued_indices;
        valued_indices.reserve(cur_node->size());
        for (int bvh_id : cur_node->prims) {
            valued_indices.emplace_back(bvh_infos[bvh_id].centroid[max_axis],
                                        bvh_id);
        }

        // Step 5: reordering the BVH info in the vector to make the segment
        // contiguous (keep around half of the bvh in lchild)
        int half_size = valued_indices.size() / 2;
        std::nth_element(
            valued_indices.begin(), valued_indices.begin() + half_size,
            valued_indices.end(),
            [](const auto &a, const auto &b) { return a.first < b.first; });

        for (int i = 0; i < half_size; i++) {
            int bvh_id = valued_indices[i].second;
            lchild_idxs.push_back(bvh_id);
            fwd_bound += bvh_infos[bvh_id].bound;
        }
        for (int i = half_size; i < valued_indices.size(); i++) {
            int bvh_id = valued_indices[i].second;
            rchild_idxs.push_back(bvh_id);
            bwd_bound += bvh_infos[bvh_id].bound;
        }
        fwd_bound ^= cur_node->bound;
        bwd_bound ^= cur_node->bound;
        float split_cost =
            traverse_cost +
            (1.f / cur_node->bound.area()) *
                (fwd_bound.area() * float(half_size) +
                 bwd_bound.area() * float(valued_indices.size() - half_size));
        if (split_cost >= node_prim_cnt && prim_num <= max_prim_node)
            fwd_bound.clear();
    }

    if (!lchild_idxs.empty() && !rchild_idxs.empty() && fwd_bound.is_valid() &&
        bwd_bound.is_valid()) {
        // in no case should the child node bound exceeds the father bound
        cur_node->release(); // release mem for non-leaf nodes
        cur_node->lchild =
            new SBVHNode(std::move(fwd_bound), std::move(lchild_idxs));
        cur_node->rchild =
            new SBVHNode(std::move(bwd_bound), std::move(rchild_idxs));
        cur_node->axis = max_axis;
    } else {
        return process_leaf();
    }
}

static int recursive_sbvh_SAH(
    const std::vector<Vec3> &points1, const std::vector<Vec3> &points2,
    const std::vector<Vec3> &points3, const std::vector<BVHInfo> &bvh_infos,
    std::vector<int> &flattened_idxs, SBVHNode *const cur_node, int depth,
    float root_area, int max_prim_node = 16, bool ref_unsplit = true) {

    auto cur_task = SBVHBuilderTask{cur_node, depth};

    // a single-threaded sbvh build function
    const auto recursive_sbvh_SAH_impl =
        [&points1, &points2, &points3, &bvh_infos, root_area, max_prim_node,
         ref_unsplit](const SBVHBuilderThreadSpan &threads,
                      const SBVHBuilderTask &task,
                      auto &&recursive_sbvh_SAH_impl) -> void {
        node_sbvh_SAH(points1, points2, points3, bvh_infos, threads, task,
                      root_area, max_prim_node, ref_unsplit);
        if (task.is_leaf())
            return;

        auto [lchild_task, rchild_task] = task.get_child_tasks();
        recursive_sbvh_SAH_impl(threads, lchild_task, recursive_sbvh_SAH_impl);
        recursive_sbvh_SAH_impl(threads, rchild_task, recursive_sbvh_SAH_impl);
    };

    // multi-threading primitives
    std::vector<SBVHBuilderThread> parallel_threads(number_of_workers - 1);
    static_assert(std::atomic<SBVHBuilderTaskKey>::is_always_lock_free);
    // estimate task_queue capacity to be the total primitive count
    atomic_queue::AtomicQueueB<SBVHBuilderTaskKey> task_queue(cur_node->size());
    std::atomic_int queued_task_count{0};

    // a multi-threaded sbvh build function
    const auto parallel_sbvh_SAH_impl =
        [&recursive_sbvh_SAH_impl, &task_queue, &queued_task_count, &points1,
         &points2, &points3, &bvh_infos, root_area, max_prim_node, ref_unsplit](
            const SBVHBuilderThreadSpan &threads, const SBVHBuilderTask &task,
            auto &&parallel_sbvh_SAH_impl) -> void {
        node_sbvh_SAH(points1, points2, points3, bvh_infos, threads, task,
                      root_area, max_prim_node, ref_unsplit);

        if (threads.can_parallelize()) {
            // if can parallelize (thread_count > 1), check the left and right
            // thread spans and run in parallel if necessary.
            if (task.is_leaf())
                return;

            // Get child tasks and thread spans
            auto [lchild_task, rchild_task] = task.get_child_tasks();
            auto [lchild_threads, rchild_threads] =
                threads.get_child_spans(lchild_task, rchild_task);

            // queue left or right tasks if their thread span is too small
            // (thread_count == 0). otherwise run the tasks in parallel.
            if (lchild_threads.should_queued()) {
                queued_task_count.fetch_add(1, std::memory_order_release);
                task_queue.push(lchild_task.get_key());
                parallel_sbvh_SAH_impl(rchild_threads, rchild_task,
                                       parallel_sbvh_SAH_impl);
            } else if (rchild_threads.should_queued()) {
                queued_task_count.fetch_add(1, std::memory_order_release);
                task_queue.push(rchild_task.get_key());
                parallel_sbvh_SAH_impl(lchild_threads, lchild_task,
                                       parallel_sbvh_SAH_impl);
            } else {
                std::future<void> rchild_future = rchild_threads.run_async([&] {
                    parallel_sbvh_SAH_impl(rchild_threads, rchild_task,
                                           parallel_sbvh_SAH_impl);
                });
                parallel_sbvh_SAH_impl(lchild_threads, lchild_task,
                                       parallel_sbvh_SAH_impl);
                rchild_future.wait();
            }
        } else {
            // if not able to parallelize (thread_count == 1), turn the thread
            // into a task queue consumer for load balance
            if (!task.is_leaf()) {
                auto [lchild_task, rchild_task] = task.get_child_tasks();
                queued_task_count.fetch_add(2, std::memory_order_release);
                task_queue.push(lchild_task.get_key());
                task_queue.push(rchild_task.get_key());
            }

            SBVHBuilderTaskKey consume_task_key;
            SBVHBuilderTask consume_task;
            // https://github.com/cameron314/concurrentqueue/blob/master/samples.md
            // refer to "Multithreaded game loop"
            while (queued_task_count.load(std::memory_order_acquire) != 0) {
                if (!task_queue.try_pop(consume_task_key))
                    continue;

                consume_task = SBVHBuilderTask::from_key(consume_task_key);
                if (consume_task.cur_node->prim_num() <
                    queue_workload_threshold) {
                    recursive_sbvh_SAH_impl(threads, consume_task,
                                            recursive_sbvh_SAH_impl);
                    queued_task_count.fetch_sub(1, std::memory_order_release);
                } else {
                    node_sbvh_SAH(points1, points2, points3, bvh_infos, threads,
                                  consume_task, root_area, max_prim_node,
                                  ref_unsplit);

                    if (consume_task.is_leaf()) {
                        queued_task_count.fetch_sub(1,
                                                    std::memory_order_release);
                    } else {
                        auto [lchild_task, rchild_task] =
                            consume_task.get_child_tasks();
                        queued_task_count.fetch_add(1,
                                                    std::memory_order_release);
                        task_queue.push(lchild_task.get_key());
                        task_queue.push(rchild_task.get_key());
                    }
                }
            }
        }
    };

    parallel_sbvh_SAH_impl(SBVHBuilderThreadSpan{parallel_threads}, cur_task,
                           parallel_sbvh_SAH_impl);

    // traverse SBVH single-threaded to flatten leaf primitives,
    // also update max_depth and node_num
    int node_num = 0;
    const auto iterate_sbvh_impl =
        [&node_num, &flattened_idxs](const SBVHBuilderTask &task,
                                     auto &&iterate_sbvh_impl) -> void {
        ++node_num;
        if (task.is_leaf()) {
            task.cur_node->base() = static_cast<int>(flattened_idxs.size());
            max_depth = std::max(max_depth, task.depth);
            for (int prim_id : task.cur_node->prims) {
                flattened_idxs.push_back(prim_id);
            }
        } else {
            auto [lchild_task, rchild_task] = task.get_child_tasks();
            iterate_sbvh_impl(lchild_task, iterate_sbvh_impl);
            iterate_sbvh_impl(rchild_task, iterate_sbvh_impl);
        }
    };

    iterate_sbvh_impl(cur_task, iterate_sbvh_impl);
    return node_num;
}

static SBVHNode *sbvh_root_start(const std::vector<Vec3> &points1,
                                 const std::vector<Vec3> &points2,
                                 const std::vector<Vec3> &points3,
                                 const Vec3 &world_min, const Vec3 &world_max,
                                 std::vector<int> &flattened_idxs,
                                 std::vector<BVHInfo> &bvh_infos, int &node_num,
                                 int max_prim_node = 16,
                                 bool ref_unsplit = true) {
    // Build BVH tree root node and start recursive tree construction
    printf("[SBVH] World min: ");
    print_vec3(world_min);
    printf("[SBVH] World max: ");
    print_vec3(world_max);
    std::vector<int> all_prims(points1.size());
    std::iota(all_prims.begin(), all_prims.end(), 0);
    SBVHNode *root_node = new SBVHNode(
        AABB(world_min, world_max, 0, points1.size()), std::move(all_prims));
    node_num = recursive_sbvh_SAH(
        points1, points2, points3, bvh_infos, flattened_idxs, root_node, 0,
        root_node->bound.area(), max_prim_node, ref_unsplit);

    return root_node;
}

template <typename ContainerTy, size_t Dim = 3>
void remap_helper_func(const std::vector<int> &flattened_idxs,
                       ContainerTy &source) {
    static constexpr int n_threads = 4;
    const size_t num_new_prims = flattened_idxs.size();
    const size_t padded_size =
        (num_new_prims + n_threads - 1) / n_threads; // workload for each thread

    ContainerTy mapped_vals;
    if constexpr (Dim == 1) {
        mapped_vals.resize(num_new_prims);
    } else {
        for (int i = 0; i < 3; i++) {
            mapped_vals[i].resize(num_new_prims);
        }
    }
#pragma omp parallel for num_threads(n_threads)
    for (int tid = 0; tid < n_threads; tid++) {
        const size_t s_pos = tid * padded_size,
                     e_pos = std::min(s_pos + padded_size, num_new_prims);
        if constexpr (Dim == 1) {
            for (size_t i = s_pos; i < e_pos; i++) {
                int index = flattened_idxs[i];
                mapped_vals[i] = source[index];
            }
        } else {
#pragma unroll
            for (int dim = 0; dim < Dim; dim++) {
                const auto &old_vec = source[dim];
                auto &new_vec = mapped_vals[dim];
                for (size_t i = s_pos; i < e_pos; i++) {
                    int index = flattened_idxs[i];
                    new_vec[i] = old_vec[index];
                }
            }
        }
    }
    source = std::move(mapped_vals);
}

void SBVHBuilder::post_process(std::vector<int> &obj_indices,
                               std::vector<int> &emitter_prims) {
    // remap all the vertices, normals, UVs and object indices for SBVH. There
    // are two major step for this: (1) reordered vertices, normals, UVs, object
    // index and sphere_flags using an multi-threading approach (or SIMD). (2)
    // Deal with the emissive primitives (remove duplication)
    size_t original_size = vertices[0].size();
    remap_helper_func(flattened_idxs, vertices);
    remap_helper_func(flattened_idxs, normals);
    remap_helper_func(flattened_idxs, uvs);
    remap_helper_func<std::vector<int>, 1>(flattened_idxs, obj_indices);
    remap_helper_func<std::vector<bool>, 1>(flattened_idxs, sphere_flags);

    const size_t num_prims = flattened_idxs.size();
    std::vector<std::vector<int>> eprim_idxs(num_emitters);
    std::vector<bool> visited(original_size, false);
    for (int i = 0; i < num_prims; i++) {
        // skip duplicated emissive primitives, if the duplicated primitives are
        // not skipped over, the emissive primitive sampling will be biased so
        // the emissive indices should be unique
        int origin_prim_id = flattened_idxs[i];
        if (visited[origin_prim_id])
            continue;
        visited[origin_prim_id] = true;

        int obj_idx = obj_indices[i] & 0x000fffff;
        const auto &object = objects[obj_idx];
        if (object.is_emitter()) {
            int emitter_idx = object.emitter_id - 1;
            eprim_idxs[emitter_idx].push_back(i);
        }
    }

    std::vector<int> e_prim_offsets;
    e_prim_offsets.push_back(0);
    for (const auto &eprim_idx : eprim_idxs) {
        e_prim_offsets.push_back(eprim_idx.size());
        for (int index : eprim_idx)
            emitter_prims.push_back(index);
    }
    std::partial_sum(e_prim_offsets.begin(), e_prim_offsets.end(),
                     e_prim_offsets.begin());
    for (ObjInfo &obj : objects) {
        if (!obj.is_emitter())
            continue;
        obj.prim_offset = e_prim_offsets[obj.emitter_id - 1];
    }
}

// Try to use two threads to build the BVH
void SBVHBuilder::build(const std::vector<int> &obj_med_idxs,
                        const Vec3 &world_min, const Vec3 &world_max,
                        std::vector<int> &obj_idxs, std::vector<float4> &nodes,
                        std::vector<CompactNode> &cache_nodes,
                        int &cache_max_level, bool ref_unsplit) {
    const auto &points1 = vertices[0], &points2 = vertices[1],
               &points3 = vertices[2];

    std::vector<PrimMappingInfo> idx_prs;
    std::vector<BVHInfo> bvh_infos;
    int node_num = 0, num_prims_all = points1.size();
    BVHBuilder::index_input(objects, sphere_flags, idx_prs, num_prims_all);
    BVHBuilder::create_bvh_info(points1, points2, points3, idx_prs,
                                obj_med_idxs, bvh_infos);

    // spatial split almost always ends up with more primitives
    flattened_idxs.reserve(num_prims_all * 2);
    SBVHNode *root_node = sbvh_root_start(points1, points2, points3, world_min,
                                          world_max, flattened_idxs, bvh_infos,
                                          node_num, max_prim_node, ref_unsplit);

    printf("[SBVH] SBVH tree max depth: %d, duplicated primitives: %lu (%lu)\n",
           max_depth, flattened_idxs.size(), points1.size());
    float total_cost =
        calculate_cost(root_node, traverse_cost, spatial_traverse_cost);
    printf("[SBVH] Traversed BVH SAH cost: %.7f, AVG: %.7f\n", total_cost,
           total_cost / static_cast<float>(bvh_infos.size()));
    calculate_tree_metrics(root_node);

    cache_max_level = std::min(std::max(max_depth - 1, 0), cache_max_level);
    nodes.reserve(node_num << 1);
    cache_nodes.reserve(1 << cache_max_level);

    recursive_linearize(root_node, nodes, cache_nodes, 0, cache_max_level);

    printf("[SBVH] Number of nodes to cache: %lu (%d)\n", cache_nodes.size(),
           cache_max_level);
    // only for debug: level_order_traverse(root_node, 8);

    obj_idxs.reserve(bvh_infos.size());
    for (BVHInfo &bvh : bvh_infos) {
        obj_idxs.emplace_back(bvh.bound.__bytes1);
    }
    delete root_node;
}
