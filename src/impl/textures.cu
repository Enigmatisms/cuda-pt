#include "hip/hip_runtime.h"
// Copyright (C) 2025 Qianyue He
//
// This program is free software: you can redistribute it and/or
// modify it under the terms of the GNU Affero General Public License
// as published by the Free Software Foundation, either
// version 3 of the License, or (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See
// the GNU Affero General Public License for more details.
//
// You should have received a copy of the GNU Affero General
// Public License along with this program. If not, see
//
//             <https://www.gnu.org/licenses/>.

/**
 * @author Qianyue He
 * @brief Texture utility function
 * @date Unknown
 */
#include "core/textures.cuh"
#include <iostream>
#include <omp.h>

#define STB_IMAGE_IMPLEMENTATION
#include "ext/stb/stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "ext/stb/stb_image_write.h"

CPT_GPU_CONST Textures c_textures;

bool save_image(const std::string &filename,
                const std::vector<unsigned char> &image_data, int width,
                int height, std::string format, const int quality) {
    if (format == "png") {
        // last parameter: line size
        return stbi_write_png(filename.c_str(), width, height, 4,
                              image_data.data(), width * 4);
    } else if (format == "jpg" || format == "jpeg") {
        // last parameter: compression quality (0 - 100)
        return stbi_write_jpg(filename.c_str(), width, height, 4,
                              image_data.data(), quality);
    } else {
        std::cerr << "Unsupported format: " << format << std::endl;
        return false;
    }
}

static bool load_image_to_float4(const std::string &filename,
                                 std::vector<float4> &out_data, int &width,
                                 int &height, float offset = 0.f,
                                 float scale = 1.f) {
    int n_channels;
    // force 4 channels
    unsigned char *data =
        stbi_load(filename.c_str(), &width, &height, &n_channels, 4);
    if (!data) {
        std::cerr << "Failed to load image: " << filename << std::endl;
        return false;
    }

    int num_pixels = width * height;
    out_data.resize(num_pixels);

#pragma omp parallel for num_threads(2)
    for (int i = 0; i < num_pixels; ++i) {
        out_data[i].x =
            (static_cast<float>(data[4 * i + 0]) / 255.0f) * scale + offset;
        out_data[i].y =
            (static_cast<float>(data[4 * i + 1]) / 255.0f) * scale + offset;
        out_data[i].z =
            (static_cast<float>(data[4 * i + 2]) / 255.0f) * scale + offset;
        out_data[i].w =
            (static_cast<float>(data[4 * i + 3]) / 255.0f) * scale + offset;
    }

    stbi_image_free(data);
    return true;
}

/**
 * @brief load two maps and composed them to a float2 map
 * note that if the second file is not presented, the second value is set to
 * 0.01
 * @param offset     Allow the function to offset the value
 * @param scale      Allow the function to scale the value
 * @param to_alpha   If true, roughness to alpha mapping will be applied (for
 * roughness map)
 */
static bool load_composed_float2(std::string file1, std::string file2,
                                 std::vector<float2> &out_data, int &width,
                                 int &height, float offset = 0.0f,
                                 float scale = 1.0f, bool to_alpha = false) {
    int n_channels, w2, h2;
    // force 4 channels
    unsigned char *data1 =
        stbi_load(file1.c_str(), &width, &height, &n_channels, 1);
    unsigned char *data2 =
        file2.length() > 1 ? stbi_load(file2.c_str(), &w2, &h2, &n_channels, 1)
                           : nullptr;

    if (!data1) {
        std::cerr << "Failed to load primary image: " << file1 << std::endl;
        return false;
    }
    if (data2) {
        if (w2 != width || h2 != height) {
            std::cerr << "Composed image size mismatch: (" << width << ", "
                      << height << "), (" << w2 << ", " << h2 << ")\n";
            return false;
        }
    }

    int num_pixels = width * height;
    out_data.resize(num_pixels);

#pragma omp parallel for num_threads(2)
    for (int i = 0; i < num_pixels; ++i) {
        float v1 = static_cast<float>(data1[i + 0]) / 255.0f,
              v2 = data2 ? static_cast<float>(data2[i + 1]) / 255.0f : v1;
        v1 = v1 * scale + offset;
        v2 = v2 * scale + offset;
        out_data[i].x = to_alpha ? roughness_to_alpha(v1) : v1;
        out_data[i].y = to_alpha ? roughness_to_alpha(v2) : v2;
    }

    stbi_image_free(data1);
    if (data2)
        stbi_image_free(data2);
    return true;
}

template <typename TexType>
static hipTextureObject_t create_texture2d(const TexType *host_data, int width,
                                            int height, TexType **d_ptr_out) {
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc<TexType>();

    TexType *d_ptr;
    size_t pitch;
    CUDA_CHECK_RETURN(
        hipMallocPitch(&d_ptr, &pitch, width * sizeof(TexType), height));

    size_t host_pitch = width * sizeof(TexType);
    CUDA_CHECK_RETURN(hipMemcpy2D(d_ptr, pitch, host_data, host_pitch,
                                   host_pitch, height, hipMemcpyHostToDevice));

    hipResourceDesc res_desc;
    memset(&res_desc, 0, sizeof(res_desc));
    res_desc.resType = hipResourceTypePitch2D;
    res_desc.res.pitch2D.devPtr = d_ptr;
    res_desc.res.pitch2D.desc = channel_desc;
    res_desc.res.pitch2D.width = width;
    res_desc.res.pitch2D.height = height;
    res_desc.res.pitch2D.pitchInBytes = pitch;

    hipTextureDesc tex_desc;
    memset(&tex_desc, 0, sizeof(tex_desc));
    tex_desc.addressMode[0] = hipAddressModeWrap;
    tex_desc.addressMode[1] = hipAddressModeWrap;
    tex_desc.filterMode = hipFilterModeLinear;
    tex_desc.readMode = hipReadModeElementType;
    tex_desc.normalizedCoords = 1;

    hipTextureObject_t tex_obj = 0;
    CUDA_CHECK_RETURN(
        hipCreateTextureObject(&tex_obj, &res_desc, &tex_desc, NULL));

    *d_ptr_out = d_ptr;
    return tex_obj;
}

template <typename TexTy>
Texture<TexTy>::Texture(std::string path, TextureType _ttype, std::string path2,
                        bool is_roughness_ior, bool is_normal_map)
    : ttype(_ttype) {
    std::vector<TexTy> host_data;
    int width = 0, height = 0;
    bool result = false;
    // Note that we don't perform type check. For example, _ttype = NORMAL_TEX,
    // while TexTy is float2, is allowed. Allowed, but the code will break down.
    // Sure, allowed, huh.
    if constexpr (std::is_same_v<TexTy, float4>) {
        result = load_image_to_float4(path, host_data, width, height,
                                      is_normal_map ? -1.f : 0.f,
                                      is_normal_map ? 2.f : 1.f);
    } else {
        if (is_roughness_ior) {
            result = load_composed_float2(path, path2, host_data, width, height,
                                          1, 1.5, true);
        } else {
            result = load_composed_float2(
                path, path2, host_data, width, height, 0, 1,
                false); // max ior: 2.5, range [1, 2.5]
        }
    }
    if (result == false) {
        std::cerr << "Texture '" << path << "' failed to load." << std::endl;
        throw std::runtime_error("Failed to load texture resources.");
    }
    _obj = create_texture2d<TexTy>(host_data.data(), width, height, &_data);
}

template <typename TexTy> void Texture<TexTy>::destroy() {
    CUDA_CHECK_RETURN(hipFree(_data));
    CUDA_CHECK_RETURN(hipDestroyTextureObject(_obj));
}

template class Texture<float2>;
template class Texture<float4>;
