// Copyright (C) 2025 Qianyue He
//
// This program is free software: you can redistribute it and/or
// modify it under the terms of the GNU Affero General Public License
// as published by the Free Software Foundation, either
// version 3 of the License, or (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See
// the GNU Affero General Public License for more details.
//
// You should have received a copy of the GNU Affero General
// Public License along with this program. If not, see
//
//             <https://www.gnu.org/licenses/>.

#include "core/scene.cuh"
#include "renderer/light_tracer.cuh"
#include "renderer/volume_pt.cuh"
#include "renderer/wf_path_tracer.cuh"

extern CPT_GPU_CONST Emitter *c_emitter[9];
extern CPT_GPU_CONST BSDF *c_material[48];

int main(int argc, char **argv) {
    CUDA_CHECK_RETURN(hipFree(nullptr)); // initialize CUDA
    if (argc < 2) {
        std::cerr << "Input file not specified. Usage: ./pt <path to xml>\n";
        exit(1);
    }
    std::string xml_path = argv[1];

    std::cout << "[SCENE] Loading scenes from '" << xml_path << "'\n";
    Scene scene(xml_path);

    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(c_material), scene.bsdfs,
                                         scene.num_bsdfs * sizeof(BSDF *)));
    CUDA_CHECK_RETURN(
        hipMemcpyToSymbol(HIP_SYMBOL(c_emitter), scene.emitters,
                           (scene.num_emitters + 1) * sizeof(Emitter *)));
#ifdef TRIANGLE_ONLY
    printf("[ATTENTION] Note that TRIANGLE_ONLY macro is defined. Please make "
           "sure there is no sphere primitive in the scene.\n");
#endif
    std::unique_ptr<TracerBase> renderer = nullptr;
    std::cout << "[RENDERER] Path tracer loaded: ";
    switch (scene.rdr_type) {
    case RendererType::MegaKernelPT: {
        renderer = std::make_unique<PathTracer<SingleTileScheduler>>(scene);
        std::cout << "\tMegakernel Path Tracing (Static Scheduler).\n";
        break;
    }
    case RendererType::WavefrontPT: {
        renderer = std::make_unique<WavefrontPathTracer>(scene);
        std::cout << "\tWavefront Path Tracing.\n";
        break;
    }
    case RendererType::MegaKernelLT: {
        renderer = std::make_unique<LightTracer>(
            scene, scene.config.spec_constraint, scene.config.bidirectional,
            scene.config.caustic_scaling);
        if (scene.config.bidirectional)
            std::cout << "\tNaive Bidirectional Megakernel Light Tracing.\n";
        else
            std::cout << "\tMegakernel Light Tracing.\n";
        break;
    }
    case RendererType::MegaKernelVPT: {
        renderer = std::make_unique<VolumePathTracer>(scene);
        std::cout << "\tVolumetric Path Tracer\n";
        break;
    }
    case RendererType::VoxelSDFPT: {
        std::cerr
            << "\tVoxelSDFPT is not implemented yet. Stay tuned. Rendering "
               "exits.\n";
        return 0;
    }
    case RendererType::AcceleratorOnly: {
        std::cout << "\tOnly building (S)BVH accelerator.\n";
        break;
    }
    case RendererType::MegaKernelPTDynamic: {
        renderer =
            std::make_unique<PathTracer<PreemptivePersistentTileScheduler>>(
                scene);
        std::cout << "\tMegakernel Path Tracing (Dynamic Scheduler).\n";
        break;
    }
    default: {
        std::cerr << "Renderer type: '" << RENDER_TYPE_STR[scene.rdr_type]
                  << "' unsupported for offline renderer.\n";
    }
    }
    if (renderer == nullptr) {
        scene.print();
        return 0;
    }
    renderer->update_camera(scene.cam);

    printf("[RENDERER] Prepare to render the scene... [%d] bounces, [%d] SPP\n",
           scene.config.md.max_depth, scene.config.spp);
    auto bytes_buffer = renderer->render(scene.config.md, scene.config.spp,
                                         scene.config.gamma_correction);

    std::string file_name = "render.png";
    if (!save_image(file_name, bytes_buffer, scene.config.width,
                    scene.config.height, "png")) {
        std::cerr << "stb::save_image() failed to output image" << std::endl;
        throw std::runtime_error("stb::save_image() fail");
    }

    printf("[IMAGE] Image saved to `%s`\n", file_name.c_str());
    scene.print();
    return 0;
}
